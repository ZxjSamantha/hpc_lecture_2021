#include "hip/hip_runtime.h"
#include <cstdio> 
#include <cstdlib>
#include <cmath>
#include <vector>
#include <chrono> 
using namespace std;

__global__ void subA(float *out, float *a, float *b, int N, int offset)
{
    for (int i=0; i < N; i++)
        for (int j=0; j < N; j++)
            out[N*i+j] = A[N*(i+offset)+j]    
}

__global__ void subB(float *out, float *a, float *b, int N, int offset, int )
{
    for (int i=0; i < N; i++)
        for (int j=0; j < N; j++)
            out[N/size*i+j] = B[N*i+j+offset]
}

int main(int argc, char** argv){
    int size, rank; 
    MPI_Init(&argc, &argv); 
    MPI_Comm_size(MPI_COMM_WORLD, &size); 
    MPI_Comm_size(MPI_COMM_WORLD, &rank); 

    const int N = 256; 

    vector<float> A(N*N);
    vector<float> B(N*N);
    vector<float> C(N*N, 0);
    vector<float> subA(N*N/size); 
    vector<float> subB(N*N/size);
    vector<float> subC(N*N/size, 0);

    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            A[N*i+j] = drand48();
            B[N*i+j] = drand48();
        }
    }
    int offset = N/size*rank; 

    for (int i=0; i<N/size; i++)
        for (int j=0; j<N; j++)
            subA[N*i+j] = A[N*(i+offset)+j]; 
    for (int i=0; i<N/size; i++)
        for (int j=0; j<N; j++)
            subB[N*i+j] = B[N*(i+offset)+j]; 
}


