
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void print(void) { // Call GPU on CPU
  printf("Hello GPU\n");
}

int main() {
  printf("Hello CPU\n");
  //print<<<1,1>>>();//Change the number 
  //foo<<<blocks, threads_per_block>>>(); 
  print<<<1,4>>>();//2048 wont work; 1024 is OK 
  hipDeviceSynchronize();
}
